#include "hip/hip_runtime.h"
#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>
#include <THC/THCAtomics.cuh>
#include <stdio.h>
#include <math.h>
#include <float.h>

using namespace at;

#define CUDA_KERNEL_LOOP(i, n)                                 \
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < (n); \
       i += blockDim.x * gridDim.x)

const int CUDA_NUM_THREADS = 1024;
const int kMaxGridNum = 65535;

inline int GET_BLOCKS(const int N)
{
  return std::min(kMaxGridNum, (N + CUDA_NUM_THREADS - 1) / CUDA_NUM_THREADS);
}

template <typename scalar_t>
__device__ scalar_t get_gradient_weight(scalar_t argmax_h, scalar_t argmax_w,
                                        const int h, const int w, const int height, const int width)
{

  if (argmax_h <= -1 || argmax_h >= height || argmax_w <= -1 || argmax_w >= width)
  {
    return 0;
  }

  int argmax_h_low = floor(argmax_h);
  int argmax_w_low = floor(argmax_w);
  int argmax_h_high = argmax_h_low + 1;
  int argmax_w_high = argmax_w_low + 1;

  scalar_t weight = 0;
  if (h == argmax_h_low && w == argmax_w_low)
    weight = (h + 1 - argmax_h) * (w + 1 - argmax_w);
  if (h == argmax_h_low && w == argmax_w_high)
    weight = (h + 1 - argmax_h) * (argmax_w + 1 - w);
  if (h == argmax_h_high && w == argmax_w_low)
    weight = (argmax_h + 1 - h) * (w + 1 - argmax_w);
  if (h == argmax_h_high && w == argmax_w_high)
    weight = (argmax_h + 1 - h) * (argmax_w + 1 - w);
  return weight;
}

template <typename scalar_t>
__global__ void im2col_gpu_kernel(const int n, const scalar_t *data_im,
                                             const int height, const int width, const int kernel_h, const int kernel_w,
                                             const int pad_h, const int pad_w, const int stride_h, const int stride_w,
                                             const int dilation_h, const int dilation_w,
                                             const int batch_size, const int num_channels,
                                             const int height_col, const int width_col,
                                             scalar_t *data_col)
{
  CUDA_KERNEL_LOOP(index, n)
  {
    const int w_col = index % width_col;
    const int h_col = (index / width_col) % height_col;
    const int b_col = (index / width_col / height_col) % batch_size;
    const int c_im = (index / width_col / height_col) / batch_size;
    const int c_col = c_im * kernel_h * kernel_w;

    const int h_in = h_col * stride_h - pad_h;
    const int w_in = w_col * stride_w - pad_w;
    scalar_t *data_col_ptr = data_col + ((c_col * batch_size + b_col) * height_col + h_col) * width_col + w_col;    
    const scalar_t *data_im_ptr = data_im + (b_col * num_channels + c_im) * height * width;

    for (int i = 0; i < kernel_h; ++i)
    {
      for (int j = 0; j < kernel_w; ++j)
      {
        const scalar_t h_im = h_in + i * dilation_h;
        const scalar_t w_im = w_in + j * dilation_w;


        if (h_im > -1 && w_im > -1 && h_im < height && w_im < width)
        {
          *data_col_ptr = data_im_ptr[int(h_im) * width + int(w_im)];
        }
        
        data_col_ptr += batch_size * height_col * width_col;
      }
    }
  }
}

void im2col(
    const at::Tensor data_im, const int channels,
    const int height, const int width, const int ksize_h, const int ksize_w,
    const int pad_h, const int pad_w, const int stride_h, const int stride_w,
    const int dilation_h, const int dilation_w, const int parallel_imgs,
    at::Tensor data_col)
{
  int height_col = (height + 2 * pad_h - (dilation_h * (ksize_h - 1) + 1)) / stride_h + 1;
  int width_col = (width + 2 * pad_w - (dilation_w * (ksize_w - 1) + 1)) / stride_w + 1;
  int num_kernels = channels * height_col * width_col * parallel_imgs;

  AT_DISPATCH_FLOATING_TYPES_AND_HALF(
      data_im.scalar_type(), "im2col_gpu", ([&] {
        const scalar_t *data_im_ = data_im.data_ptr<scalar_t>();
        scalar_t *data_col_ = data_col.data_ptr<scalar_t>();

        im2col_gpu_kernel<<<GET_BLOCKS(num_kernels), 
                                       CUDA_NUM_THREADS, 0, 
                                       at::cuda::getCurrentCUDAStream()>>>(
            num_kernels, data_im_, height, width, ksize_h, ksize_w,
            pad_h, pad_w, stride_h, stride_w, dilation_h, dilation_w,
            parallel_imgs, channels,
            height_col, width_col, data_col_);
      }));

  hipError_t err = hipGetLastError();
  if (err != hipSuccess)
  {
    printf("error in im2col: %s\n", hipGetErrorString(err));
  }
}

template <typename scalar_t>
__global__ void col2im_gpu_kernel(
    const int n, const scalar_t *data_col,
    const int channels, const int height, const int width,
    const int kernel_h, const int kernel_w,
    const int pad_h, const int pad_w,
    const int stride_h, const int stride_w,
    const int dilation_h, const int dilation_w,
    const int batch_size,
    const int height_col, const int width_col,
    scalar_t *grad_im)
{
  CUDA_KERNEL_LOOP(index, n)
  {
    const int j = (index / width_col / height_col / batch_size) % kernel_w;
    const int i = (index / width_col / height_col / batch_size / kernel_w) % kernel_h;
    const int c = index / width_col / height_col / batch_size / kernel_w / kernel_h;

    int w_out = index % width_col;
    int h_out = (index / width_col) % height_col;
    int b = (index / width_col / height_col) % batch_size;
    int w_in = w_out * stride_w - pad_w;
    int h_in = h_out * stride_h - pad_h;

    const scalar_t cur_inv_h_data = h_in + i * dilation_h;
    const scalar_t cur_inv_w_data = w_in + j * dilation_w;

    const scalar_t cur_top_grad = data_col[index];
    const int cur_h = (int)cur_inv_h_data;
    const int cur_w = (int)cur_inv_w_data;
    for (int dy = -2; dy <= 2; dy++)
    {
      for (int dx = -2; dx <= 2; dx++)
      {
        if (cur_h + dy >= 0 && cur_h + dy < height &&
            cur_w + dx >= 0 && cur_w + dx < width &&
            abs(cur_inv_h_data - (cur_h + dy)) < 1 &&
            abs(cur_inv_w_data - (cur_w + dx)) < 1)
        {
          int cur_bottom_grad_pos = ((b * channels + c) * height + cur_h + dy) * width + cur_w + dx;
          scalar_t weight = get_gradient_weight(cur_inv_h_data, cur_inv_w_data, cur_h + dy, cur_w + dx, height, width);
          atomicAdd(grad_im + cur_bottom_grad_pos, weight * cur_top_grad);
        }
      }
    }
  }
}

void col2im(
    const at::Tensor data_col, const int channels,
    const int height, const int width, const int ksize_h,
    const int ksize_w, const int pad_h, const int pad_w,
    const int stride_h, const int stride_w,
    const int dilation_h, const int dilation_w,
    const int parallel_imgs,
    at::Tensor grad_im)

{
  int height_col = (height + 2 * pad_h - (dilation_h * (ksize_h - 1) + 1)) / stride_h + 1;
  int width_col = (width + 2 * pad_w - (dilation_w * (ksize_w - 1) + 1)) / stride_w + 1;
  int num_kernels = channels * ksize_h * ksize_w * height_col * width_col * parallel_imgs;

  AT_DISPATCH_FLOATING_TYPES_AND_HALF(
      data_col.scalar_type(), "col2im_gpu", ([&] {
        const scalar_t *data_col_ = data_col.data_ptr<scalar_t>();
        scalar_t *grad_im_ = grad_im.data_ptr<scalar_t>();

        col2im_gpu_kernel<<<GET_BLOCKS(num_kernels), CUDA_NUM_THREADS, 0, at::cuda::getCurrentCUDAStream()>>>(
            num_kernels, data_col_, channels, height, width, ksize_h,
            ksize_w, pad_h, pad_w, stride_h, stride_w,
            dilation_h, dilation_w,
            parallel_imgs, height_col, width_col, grad_im_);
      }));

  hipError_t err = hipGetLastError();
  if (err != hipSuccess)
  {
    printf("error in col2im: %s\n", hipGetErrorString(err));
  }
}